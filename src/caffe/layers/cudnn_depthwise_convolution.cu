
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_depthwise_convolution.hpp"

namespace caffe {

	__global__ void sync_depthwise_conv_groups() { }

	template <typename Dtype>
	void CuDNNDepthwiseConvolutionLayer<Dtype>::Forward_gpu(
		const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
		//printf("##################################################### %d\r\n", top[0]->num());
		//printf("##################################################### %d\r\n", top[0]->channels());
		//printf("##################################################### %d\r\n", top[0]->height());
		//printf("##################################################### %d\r\n", top[0]->width());
		//printf("##################################################### %d\r\n", top[0]->gpu_data());
		//printf("##################################################### before cudnn depthwise\r\n");
		const Dtype* weight = this->blobs_[0]->gpu_data();
		for (int i = 0; i < bottom.size(); ++i) {
			const Dtype* bottom_data = bottom[i]->gpu_data();
			Dtype* top_data = top[i]->mutable_gpu_data();
            //printf("##################################################### test cudnn depthwise\r\n");
			// Forward through cuDNN in parallel over groups.
			//for (int g = 0; g < this->group_; g++) {
			{
				// Filters.
				CUDNN_CHECK(cudnnConvolutionForward(handle_,
					cudnn::dataType<Dtype>::one,
					bottom_descs_[i],
					bottom_data + bottom_offset_,
					filter_desc_,
					weight + this->weight_offset_,
					conv_descs_[i],
					fwd_algo_[i],
					workspace[0],
					workspace_fwd_sizes_[i],
					cudnn::dataType<Dtype>::zero,
					top_descs_[i], 
					top_data + top_offset_));				

				// Bias.
				if (this->bias_term_) {
					const Dtype* bias_data = this->blobs_[1]->gpu_data();
					CUDNN_CHECK(cudnnAddTensor(handle_,
						cudnn::dataType<Dtype>::one,
						bias_desc_, bias_data + bias_offset_,
						cudnn::dataType<Dtype>::one,
						top_descs_[i], top_data + top_offset_));
				}
			}

			// Synchronize the work across groups, each of which went into its own
			// stream, by launching an empty kernel into the default (null) stream.
			// NOLINT_NEXT_LINE(whitespace/operators)
			//sync_depthwise_conv_groups<<<1, 1>>>();
		}
		//printf("##################################################### after cudnn depthwise\r\n");
	}

	template <typename Dtype>
	void CuDNNDepthwiseConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
		const Dtype* weight = NULL;
		Dtype* weight_diff = NULL;
		if (this->param_propagate_down_[0]) {
			weight = this->blobs_[0]->gpu_data();
			weight_diff = this->blobs_[0]->mutable_gpu_diff();
		}
		Dtype* bias_diff = NULL;
		if (this->bias_term_ && this->param_propagate_down_[1]) {
			bias_diff = this->blobs_[1]->mutable_gpu_diff();
		}
		for (int i = 0; i < top.size(); ++i) {
			const Dtype* top_diff = top[i]->gpu_diff();
			// Backward through cuDNN in parallel over groups and gradients.
			//for (int g = 0; g < this->group_; g++) {
			{
				// Gradient w.r.t. bias.
				if (this->bias_term_ && this->param_propagate_down_[1]) {
					CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_,
						cudnn::dataType<Dtype>::one,
						top_descs_[i], top_diff + top_offset_,
						cudnn::dataType<Dtype>::one,
						bias_desc_, bias_diff + bias_offset_));
				}

				// Gradient w.r.t. weights.
				if (this->param_propagate_down_[0]) {
					const Dtype* bottom_data = bottom[i]->gpu_data();
					CUDNN_CHECK(cudnnConvolutionBackwardFilter(
						handle_,
						cudnn::dataType<Dtype>::one,
						bottom_descs_[i], bottom_data + bottom_offset_,
						top_descs_[i], top_diff + top_offset_,
						conv_descs_[i],
						bwd_filter_algo_[i], workspace[0],
						workspace_bwd_filter_sizes_[i],
						cudnn::dataType<Dtype>::one,
						filter_desc_, weight_diff + this->weight_offset_));
				}

				// Gradient w.r.t. bottom data.
				if (propagate_down[i]) {
					if (weight == NULL) {
						weight = this->blobs_[0]->gpu_data();
					}
					Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
					CUDNN_CHECK(cudnnConvolutionBackwardData(
						handle_,
						cudnn::dataType<Dtype>::one,
						filter_desc_, weight + this->weight_offset_,
						top_descs_[i], top_diff + top_offset_,
						conv_descs_[i],
						bwd_data_algo_[i], workspace[0],
						workspace_bwd_data_sizes_[i],
						cudnn::dataType<Dtype>::zero,
						bottom_descs_[i], bottom_diff + bottom_offset_));
				}
			}

			// Synchronize the work across groups, each of which went into its own
			// stream, by launching an empty kernel into the default (null) stream.
			// NOLINT_NEXT_LINE(whitespace/operators)
			//sync_conv_groups << <1, 1 >> >();
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(CuDNNDepthwiseConvolutionLayer);

}  // namespace caffe
#endif

